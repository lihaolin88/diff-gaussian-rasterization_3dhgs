#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
// __device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
// {
// 	// The following models the steps outlined by equations 29
// 	// and 31 in "EWA Splatting" (Zwicker et al., 2002).
// 	// Additionally considers aspect / scaling of viewport.
// 	// Transposes used to account for row-/column-major conventions.
// 	float3 t = transformPoint4x3(mean, viewmatrix);
//
// 	const float limx = 1.3f * tan_fovx;
// 	const float limy = 1.3f * tan_fovy;
// 	const float txtz = t.x / t.z;
// 	const float tytz = t.y / t.z;
// 	t.x = min(limx, max(-limx, txtz)) * t.z;
// 	t.y = min(limy, max(-limy, tytz)) * t.z;
//
// 	glm::mat3 J = glm::mat3(
// 		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
// 		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
// 		0, 0, 0);
//
// 	glm::mat3 W = glm::mat3(
// 		viewmatrix[0], viewmatrix[4], viewmatrix[8],
// 		viewmatrix[1], viewmatrix[5], viewmatrix[9],
// 		viewmatrix[2], viewmatrix[6], viewmatrix[10]);
//
// 	glm::mat3 T = W * J;
//
// 	glm::mat3 Vrk = glm::mat3(
// 		cov3D[0], cov3D[1], cov3D[2],
// 		cov3D[1], cov3D[3], cov3D[4],
// 		cov3D[2], cov3D[4], cov3D[5]);
//
// 	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;
//
// 	// Apply low-pass filter: every Gaussian should be at least
// 	// one pixel wide/high. Discard 3rd row and column.
// 	cov[0][0] += 0.3f;
// 	cov[1][1] += 0.3f;
// 	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
// }
__device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002).
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	float3 t = transformPoint4x3(mean, viewmatrix);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

// 	glm::mat3 W = glm::mat3(
// 		viewmatrix[0], viewmatrix[4], viewmatrix[8],
// 		viewmatrix[1], viewmatrix[5], viewmatrix[9],
// 		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

//     glm::mat3 T = W*J;
	glm::mat3 T = J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Apply low-pass filter: every Gaussian should be at least
	// one pixel wide/high. Discard 3rd row and column.
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}


__device__ float6 computeCov2D_halfgs(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, float6 view_cov3D, const float* viewmatrix)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002).
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	// Change J for half gaussian
	float3 t = transformPoint4x3(mean, viewmatrix);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	glm::mat3 J = glm::mat3(
		(focal_x / t.z), 0.0, (-(focal_x * t.x) / (t.z * t.z)),
		0.0, (focal_y / t.z), (-(focal_y * t.y) / (t.z * t.z)),
		0, 0, (focal_y / t.z));

	glm::mat3 W = glm::mat3(
		(viewmatrix[0]), (viewmatrix[4]), (viewmatrix[8]),
		(viewmatrix[1]), (viewmatrix[5]), (viewmatrix[9]),
		(viewmatrix[2]), (viewmatrix[6]), (viewmatrix[10]));

	glm::mat3 T = W * J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

    glm::mat3 rot_cov = glm::transpose(W) * glm::transpose(Vrk) * W;

    view_cov3D.x = rot_cov[0][0];
    view_cov3D.y = rot_cov[0][1];
    view_cov3D.z = rot_cov[1][1];
    view_cov3D.w = rot_cov[0][2];
    view_cov3D.u = rot_cov[1][2];
    view_cov3D.v = rot_cov[2][2];

//     glm::mat3 cov = glm::transpose(J) * glm::transpose(rot_cov) * J;


	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Apply low-pass filter: every Gaussian should be at least
	// one pixel wide/high. Discard 3rd row and column.
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	cov[2][2] += 0.3f;
	return {cov[0][0], cov[0][1], cov[1][1], cov[0][2], cov[1][2], cov[2][2]};//{ float(cov[0][0]), float(cov[0][1]), float(cov[1][1]), float(cov[0][2]), float(cov[1][2]), float(cov[2][2])};
}

// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
	// Create scaling matrix
	glm::mat3 S = glm::mat3(1.0f);
	S[0][0] = mod * scale.x;
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	glm::mat3 S_inv = glm::mat3(1.0f);
	S_inv[0][0] = 1/S[0][0];
	S_inv[1][1] = 1/S[1][1];
	S_inv[2][2] = 1/S[2][2];

	// Normalize quaternion to get valid rotation
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// Compute rotation matrix from quaternion
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 M = S * R;

	//glm::mat3 M_inv = S_inv * R;

	// Compute 3D world covariance matrix Sigma
	glm::mat3 Sigma = glm::transpose(M) * M;
	//glm::mat3 Sigma_inv = glm::transpose(M_inv) * M_inv;


	// Covariance is symmetric, only store upper right
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];

    //float6 cov3D_inv;

    //glm::mat3 test = Sigma*Sigma_inv - glm::mat3(1.0f);
    //float sum_result =fabs(test[0][0])+fabs(test[0][1])+fabs(test[0][2]) + fabs(test[1][0])+fabs(test[1][1])+fabs(test[1][2])+fabs(test[2][0])+fabs(test[2][1])+fabs(test[2][2]);

	//cov3D_inv.x = sum_result >= 0.01f?0.f:Sigma_inv[0][0];
	//cov3D_inv.y = sum_result >= 0.01f?0.f:Sigma_inv[0][1];
	//cov3D_inv.z = sum_result >= 0.01f?0.f:Sigma_inv[0][2];
	//cov3D_inv.w = sum_result >= 0.01f?0.f:Sigma_inv[1][1];
	//cov3D_inv.u = sum_result >= 0.01f?0.f:Sigma_inv[1][2];
	//cov3D_inv.v = sum_result >= 0.01f?0.f:Sigma_inv[2][2];

//     glm::mat3 test = Sigma*Sigma_inv;
// //     if(test[0][0]+test[1][1]+test[2][2]>3.1 || test[0][0]+test[1][1]+test[2][2]<2.9)
// //     {return cov3D_inv;} //save all 0 if the gaussian is too small, then inv value will be too large
//
// 	cov3D_inv.x = (test[0][0]+test[1][1]+test[2][2]>3.01 || test[0][0]+test[1][1]+test[2][2]<2.99)?0.f:Sigma_inv[0][0];
// 	cov3D_inv.y = (test[0][0]+test[1][1]+test[2][2]>3.01 || test[0][0]+test[1][1]+test[2][2]<2.99)?0.f:Sigma_inv[0][1];
// 	cov3D_inv.z = (test[0][0]+test[1][1]+test[2][2]>3.01 || test[0][0]+test[1][1]+test[2][2]<2.99)?0.f:Sigma_inv[0][2];
// 	cov3D_inv.w = (test[0][0]+test[1][1]+test[2][2]>3.01 || test[0][0]+test[1][1]+test[2][2]<2.99)?0.f:Sigma_inv[1][1];
// 	cov3D_inv.u = (test[0][0]+test[1][1]+test[2][2]>3.01 || test[0][0]+test[1][1]+test[2][2]<2.99)?0.f:Sigma_inv[1][2];
// 	cov3D_inv.v = (test[0][0]+test[1][1]+test[2][2]>3.01 || test[0][0]+test[1][1]+test[2][2]<2.99)?0.f:Sigma_inv[2][2];

// 	cov3D_inv.x = Sigma_inv[0][0];
// 	cov3D_inv.y = Sigma_inv[0][1];
// 	cov3D_inv.z = Sigma_inv[0][2];
// 	cov3D_inv.w = Sigma_inv[1][1];
// 	cov3D_inv.u = Sigma_inv[1][2];
// 	cov3D_inv.v = Sigma_inv[2][2];

	 //////

//     {printf("wrong,");}
    //{printf("inverse is not very accurate, %f, result is:%f\n",cov3D_inv.x, test[0][0]+test[1][1]+test[2][2]);}
//     printf("test inv: %f,%f,%f;%f,%f,%f,%f\n",test[0][0],test[1][1],test[2][2],test[0][1],test[0][2],test[1][2],cov3D_inv.x);
	//////

	//return cov3D_inv;
}

// __device__ void generate_type(float3 cov, float op1, float op2, float& type, float3 normal, float x_term, float y_term)
// {
//     float diagional; //determine if the ellipsoid is on diagional direction or not, in diagional is 1, anti-dia is 0
//     float leftright; //determine if op1 is on left or right, left is 1, right is 0
//
//     float full = (fabsf(op1-op2)<0.0039)?0.f:1.f; //if type is 0, then we use the whole gaussian
//     float lambda = ((cov.x+cov.z) + sqrtf((cov.x+cov.z)*(cov.x+cov.z) - 4*(cov.x*cov.z-cov.y*cov.y)))/2;
//
//     float2 vec = {lambda - cov.z - cov.y, cov.x - lambda + cov.y}; //find the long axis direction
//     float op_small = (op1-op2)>0.f?0.0f:1.0f; // p_vec will direct to op1, if op1 is not small, then we use these part, else the other part
//     //find the angle between vec and [0,1], which will tell us diagional
//     diagional = vec.x*vec.y<0.0f ? 1.0f : 0.0f; //use mean as center, vector in diagional(1,4) or in anti-diagional(2,3)
//     float temp_left_right = ((normal.x*1.4142135f+x_term)*vec.x+(normal.y*1.4142135f+y_term)*vec.y) > 0.f? vec.x:-vec.x;
//     leftright = temp_left_right<0 ? op_small-0.f:1.f-op_small;
// //     leftright = p_vec<0.0f ? 1.0f-op_small : op_small-0.0f; // left part is 1, right is 0
//
//     type = full*(diagional*1.0f*leftright + (1.0f-diagional)*2.0f*(1.0f-leftright) + (1.0f-diagional)*3.0f*leftright + diagional*4.0f*(1.0f-leftright));
// }

// __device__ void generate_type(float3 cov, float3 cov_small, float op1, float op2, float& type, float3 normal, float x_term, float y_term, float3 conic)
// {
//     float diagional; //determine if the ellipsoid is on diagional direction or not, in diagional is 1, anti-dia is 0
//     float leftright; //determine if op1 is on left or right, left is 1, right is 0
//
//     float full = (fabsf(op1-op2)<0.0039)?0.f:1.f; //if type is 0, then we use the whole gaussian
//     float lambda = ((cov.x+cov.z) + sqrtf((cov.x+cov.z)*(cov.x+cov.z) - 4*(cov.x*cov.z-cov.y*cov.y)))/2;
//
//     float lambda_small = ((cov_small.x+cov_small.z) + sqrtf((cov_small.x+cov_small.z)*(cov_small.x+cov_small.z) - 4*(cov_small.x*cov_small.z-cov_small.y*cov_small.y)))/2;
//
//     float2 vec = {lambda - cov.z - cov.y, cov.x - lambda + cov.y}; //find the long axis direction
//
//     float2 vec_small = {lambda_small - cov_small.z - cov_small.y, cov_small.x - lambda_small + cov_small.y}; //find the long axis direction
//     float2 vec_small_short = {-vec_small.y, vec_small.x};
//
//     float op_small = (op1-op2)>0.f?0.0f:1.0f; // p_vec will direct to op1, if op1 is not small, then we use these part, else the other part
//     //find the angle between vec and [0,1], which will tell us diagional
//     //diagional = vec.x*vec.y<0.0f ? 1.0f : 0.0f; //use mean as center, vector in diagional(1,4) or in anti-diagional(2,3)
//
//     float change= ((normal.x*1.4142135f+x_term)*vec_small_short.x+(normal.y*1.4142135f+y_term)*vec_small_short.y) > 0.f?1.0f:0.f;
//     change = change<0 ? op_small-0.f:1.f-op_small;
// //     vec_small_short.x = change==1.f? vec_small_short.x:-vec_small_short.x;
// //     vec_small_short.y = change==1.f? vec_small_short.y:-vec_small_short.y;
//     float temp_left_right= ((normal.x*1.4142135f+x_term)*vec.x+(normal.y*1.4142135f+y_term)*vec.y) > 0.f?1.0f:0.f;
//     leftright = temp_left_right<0 ? op_small-0.f:1.f-op_small;
//
//
// //     leftright = p_vec<0.0f ? 1.0f-op_small : op_small-0.0f; // left part is 1, right is 0
//     //generate diagional
//     float cx = conic.x;
//     float cy = conic.y;
//     float cz = conic.z;
//     float det2 = 1/sqrtf(cx*cy*cy - 2*cx*cy*cy + cz*cx*cx);
//     float2 point3 = {3*cy*det2, -3*cx*det2};
//     float2 point1 = {-3*cy*det2, 3*cx*det2};
//     det2 = 1/sqrtf(cx*cz*cz - 2*cz*cy*cy + cz*cy*cy);
//     float2 point2 = {3*cz*det2, -3*cy*det2};
//     float2 point4 = {-3*cz*det2, 3*cy*det2};
//
//     float2 vertical_point4 = {point4.y, -point4.x};
//     float2 vertical_point2 = {point2.y, -point2.x};
//
// //     float temp_left_right = ;
//
//     diagional = ((point2.x*vec_small.y-point2.y*vec_small.x)>0 && (point1.x*vec_small.y-point1.y*vec_small.x)<0) || ((point4.x*vec_small.y-point4.y*vec_small.x)<0 && (point3.x*vec_small.y-point3.y*vec_small.x)>0);
//
// //     leftright = ((vertical_point2.x*vec_small_short.y-vertical_point2.y*vec_small_short.x)<0 && (vertical_point4.x*vec_small_short.y-vertical_point4.y*vec_small_short.x)>0) ? 1.f:0.f;;
//
//     type = full*(diagional*1.0f*leftright + (1.0f-diagional)*2.0f*(1.0f-leftright) + (1.0f-diagional)*3.0f*leftright + diagional*4.0f*(1.0f-leftright));
// }

__device__ void generate_type(float3 cov, float3 cov_small, float op1, float op2, float& type, float3 normal, float x_term, float y_term, float3 conic)
{
    //float diagional; //determine if the ellipsoid is on diagional direction or not, in diagional is 1, anti-dia is 0
    //float leftright; //determine if op1 is on left or right, left is 1, right is 0

    float full = (fabsf(op1-op2)<0.0039)?0.f:1.f; //if type is 0, then we use the whole gaussian
    //float lambda = ((cov.x+cov.z) + sqrtf((cov.x+cov.z)*(cov.x+cov.z) - 4*(cov.x*cov.z-cov.y*cov.y)))/2;

    //float2 vec = {lambda - cov.z - cov.y, cov.x - lambda + cov.y}; //find the long axis direction

//     float op_small = (op1-op2)>0.f?0.0f:1.0f; // p_vec will direct to op1, if op1 is not small, then we use these part, else the other part
    //find the angle between vec and [0,1], which will tell us diagional
    //diagional = vec.x*vec.y<0.0f ? 1.0f : 0.0f; //use mean as center, vector in diagional(1,4) or in anti-diagional(2,3)
    float op_inverse = op1 > op2 ? 1.0f: 0.0f;

    float cx = conic.x;
    float cy = conic.y;
    float cz = conic.z;
    float det2 = 1/sqrtf(cx*cy*cy - 2*cx*cy*cy + cz*cx*cx);
    float2 point3 = {cy*det2, -cx*det2};
    float2 point1 = {-cy*det2, cx*det2};
    det2 = 1/sqrtf(cx*cz*cz - 2*cz*cy*cy + cz*cy*cy);
    float2 point2 = {cz*det2, -cy*det2};
    float2 point4 = {-cz*det2, cy*det2};

    float part1 = (normal.x*1.4142135f+x_term);
    float part2 = (normal.y*1.4142135f+y_term);
    float point1_large= (part1*point1.x+part2*point1.y) > 0.f?1.0f-op_inverse:0.f+op_inverse;
    float point2_large= (part1*point2.x+part2*point2.y) > 0.f?1.0f-op_inverse:0.f+op_inverse;
    float point3_large= 1.f - point1_large;//(part1*point3.x+part2*point3.y) > 0.f?1.0f-op_inverse:0.f+op_inverse;
    float point4_large= 1.f - point2_large;//(part1*point4.x+part2*point4.y) > 0.f?1.0f-op_inverse:0.f+op_inverse;

    type = full*(point1_large*point4_large*(1-point2_large)*(1-point3_large)*1.f + point1_large*point2_large*(1-point4_large)*(1-point3_large)*2.f + point3_large*point4_large*(1-point2_large)*(1-point1_large)*3.f+point2_large*point3_large*(1-point1_large)*(1-point4_large)*4.f);
//     type = full*(diagional*1.0f*leftright + (1.0f-diagional)*2.0f*(1.0f-leftright) + (1.0f-diagional)*3.0f*leftright + diagional*4.0f*(1.0f-leftright));
}


//use large cov3D and normal to generate small cov
// __device__ void calculate_small(const float* cov_input, const float* viewmatrix, float3 n, float* cov_new_small) {
//     // 将输入的 float 类型直接用于 float 运算
//     glm::vec3 normal(n.x, n.y, n.z);
//     normal = glm::normalize(normal);
//
//     // 使用 float 类型的 3x3 矩阵
//     glm::mat3 W = glm::mat3(
//         viewmatrix[0], viewmatrix[4], viewmatrix[8],
//         viewmatrix[1], viewmatrix[5], viewmatrix[9],
//         viewmatrix[2], viewmatrix[6], viewmatrix[10]);
//
//     // 声明一个 3x3 的 float 类型的协方差矩阵
//     glm::mat3 cov;
//     cov[0][0] = cov_input[0]; // cov(0,0)
//     cov[0][1] = cov[1][0] = cov_input[1]; // cov(0,1) and cov(1,0)
//     cov[0][2] = cov[2][0] = cov_input[2]; // cov(0,2) and cov(2,0)
//     cov[1][1] = cov_input[3]; // cov(1,1)
//     cov[1][2] = cov[2][1] = cov_input[4]; // cov(1,2) and cov(2,1)
//     cov[2][2] = cov_input[5]; // cov(2,2)
//
//     // 执行矩阵变换
//     cov = glm::transpose(W) * cov * W;
//
//     // 使用 float 类型的旋转矩阵
//     glm::vec3 v1, v2;
//
//     if (normal.x == 0.0f && normal.y == 0.0f) {
//         // 如果 n.x 和 n.y 为 0，设置 v1 和 v2
//         v1 = glm::vec3(1.0f, 0.0f, 0.0f);
//         v2 = glm::vec3(0.0f, 1.0f, 0.0f);
//     } else {
//         // 正交化得到 v1 和 v2
//         v1 = glm::normalize(glm::vec3(normal.y, -normal.x, 0.0f));
//         v2 = glm::normalize(glm::cross(normal, v1));
//     }
//
//     glm::mat3 R_transform(v1, v2, normal);
//     glm::mat3 R_transform_T = glm::transpose(R_transform);
//
//     // 计算 cov_transformed 和 cov_inv2，使用 float 类型
//     glm::mat3 cov_transformed = R_transform_T * cov * R_transform;
//     float a = cov_transformed[0][0];  // cov_transformed(0,0)
//     float b = cov_transformed[0][1];  // cov_transformed(0,1) == cov_transformed(1,0)
//     float c = cov_transformed[1][1];  // cov_transformed(1,1)
//     float d = cov_transformed[0][2];  // cov_transformed(0,2) == cov_transformed(2,0)
//     float e = cov_transformed[1][2];  // cov_transformed(1,2) == cov_transformed(2,1)
//     float f = cov_transformed[2][2];  // cov_transformed(2,2)
//
//     // Schur补公式: 计算 A_{11} - B * C^{-1} * B^T
//     float inv_f = 1.0f / f;  // C的逆: 1/f
//     float B_C_inv_B_T_00 = (d * d) * inv_f;
//     float B_C_inv_B_T_01 = (d * e) * inv_f;
//     float B_C_inv_B_T_11 = (e * e) * inv_f;
//
//     // A_{11} - B * C^{-1} * B^T
//     float A11_00 = a - B_C_inv_B_T_00;
//     float A11_01 = b - B_C_inv_B_T_01;
//     float A11_11 = c - B_C_inv_B_T_11;
//
//     // 计算2x2矩阵的行列式
//     float det_A11 = A11_00 * A11_11 - A11_01 * A11_01;
//
//     // 计算逆矩阵的四个元素
//     float inv_A11_00 = A11_11 / det_A11;
//     float inv_A11_01 = -A11_01 / det_A11;
//     float inv_A11_11 = A11_00 / det_A11;
//
//     glm::mat2 twoD_mat(inv_A11_00, inv_A11_01, inv_A11_01, inv_A11_11);
//
//     glm::vec2 eig_val;
//     glm::mat2 eig_vec;
//
//     // 计算 2x2 矩阵的特征值
//     float trace = twoD_mat[0][0] + twoD_mat[1][1];
//     float det = twoD_mat[0][0] * twoD_mat[1][1] - twoD_mat[0][1] * twoD_mat[1][0];
//     float disc = sqrtf(trace * trace - 4.0f * det);
//
//     eig_val[0] = (trace + disc) / 2.0f;
//     eig_val[1] = (trace - disc) / 2.0f;
//
//     // 计算特征向量
//     eig_vec[0][0] = (twoD_mat[0][0] - eig_val[0]) != 0.0f ? -(twoD_mat[0][1] / (twoD_mat[0][0] - eig_val[0])) : 1.0f;
//     eig_vec[1][0] = (twoD_mat[0][0] - eig_val[0]) != 0.0f ? 1.0f : 0.0f;
//
//     eig_vec[0][1] = (twoD_mat[0][0] - eig_val[1]) != 0.0f ? -(twoD_mat[0][1] / (twoD_mat[0][0] - eig_val[1])) : 1.0f;
//     eig_vec[1][1] = (twoD_mat[0][0] - eig_val[1]) != 0.0f ? 1.0f : 0.0f;
//
//     // 归一化特征向量
//     eig_vec[1][0] = 1.0f / sqrtf(eig_vec[0][0] * eig_vec[0][0] + 1.0f);
//     eig_vec[0][0] *= eig_vec[1][0];
//     eig_vec[1][1] = 1.0f / sqrtf(eig_vec[0][1] * eig_vec[0][1] + 1.0f);
//     eig_vec[0][1] *= eig_vec[1][1];
//
//     // 将特征向量转换为 3D
//     glm::vec3 v1_2d(eig_vec[0][0], eig_vec[1][0], 0.0f);
//     glm::vec3 v2_2d(eig_vec[0][1], eig_vec[1][1], 0.0f);
//
//     // 生成 3D 特征向量矩阵
//     glm::mat3 v_3d(v1_2d, v2_2d, glm::vec3(0.0f, 0.0f, 1.0f));
//
//     // 构建特征值矩阵
//     glm::mat3 lam_mat(0.0f);
//     lam_mat[0][0] = 1.0f / eig_val[0];
//     lam_mat[1][1] = 1.0f / eig_val[1];
//     lam_mat[2][2] = 1.0f / (1000.0f * fmaxf(eig_val[0], eig_val[1]));
//
//     // 计算新的协方差矩阵 cov_new_3d
//     glm::mat3 cov_new_3d = v_3d * lam_mat * glm::transpose(v_3d);
//
//     // 将其转换回摄像机坐标系
//     cov_new_3d = R_transform * cov_new_3d * R_transform_T;
//
//     // 最终结果存储为 float32
//     cov_new_small[0] = cov_new_3d[0][0];
//     cov_new_small[1] = cov_new_3d[0][1];
//     cov_new_small[2] = cov_new_3d[0][2];
//     cov_new_small[3] = cov_new_3d[1][1];
//     cov_new_small[4] = cov_new_3d[1][2];
//     cov_new_small[5] = cov_new_3d[2][2];
// }
// __device__ void calculate_small(float6 cov_input, const float* viewmatrix, float3 normal, float* cov_new_small) {
//     // 如果法向量 n 的 x 和 y 分量都为 0，生成标准基向量
//     glm::vec3 v1, v2;
//
//     glm::vec3 n(normal.x, normal.y, normal.z);
//     n = glm::normalize(n);
//
//     glm::mat3 W = glm::mat3(
//         viewmatrix[0], viewmatrix[4], viewmatrix[8],
//         viewmatrix[1], viewmatrix[5], viewmatrix[9],
//         viewmatrix[2], viewmatrix[6], viewmatrix[10]);
//
//     // 声明一个 3x3 的 float 类型的协方差矩阵
//     glm::mat3 cov;
//     cov[0][0] = cov_input.x; // cov(0,0)
//     cov[0][1] = cov[1][0] = cov_input.y; // cov(0,1) and cov(1,0)
//     cov[0][2] = cov[2][0] = cov_input.z; // cov(0,2) and cov(2,0)
//     cov[1][1] = cov_input.w; // cov(1,1)
//     cov[1][2] = cov[2][1] = cov_input.u; // cov(1,2) and cov(2,1)
//     cov[2][2] = cov_input.v; // cov(2,2)
//
//     cov = glm::transpose(W) * cov * W;
//
//     // 如果 n 的 x 和 y 分量都为 0，则沿 z 轴对齐
//     if (n.x == 0.0f && n.y == 0.0f) {
//         v1 = glm::vec3(1.0f, 0.0f, 0.0f);
//         v2 = glm::vec3(0.0f, 1.0f, 0.0f);
//         //n = glm::normalize(glm::vec3(0.0f, 0.0f, n.z));  // 保证法向量沿 z 轴方向
//     } else {
//         // 归一化法向量 n
//         // 生成与 n 正交的 v1 和 v2 向量
//         v1 = glm::normalize(glm::vec3(n.y, -n.x, 0.0f));  // 创建垂直于 n 的 v1
//         v2 = glm::normalize(glm::cross(n, v1));            // 通过叉乘计算 v2
//     }
// //     if (fabsf(glm::dot(v1, v2))>0.00001f ||fabsf(glm::dot(n, v2))>0.00001f||fabsf(glm::dot(n, v1))>0.00001f) printf("n not correct is：[%f,%f,%f]\n",n.x, n.y, n.z);
//
//     // 构建基变换矩阵
//     glm::mat3 basis(v1, v2, n);
//     glm::mat3 R_transform = glm::transpose(basis);  // R_transform 是基变换矩阵的转置
//
//     // 将输入协方差矩阵从 float* 转换为 glm::mat3
// //     glm::mat3 cov_mat(
// //         cov[0], cov[1], cov[2],
// //         cov[3], cov[4], cov[5],
// //         cov[6], cov[7], cov[8]);
//
//     // 对协方差矩阵进行基变换
//     glm::mat3 cov_transformed = R_transform * cov * basis;
//
//     // 提取二维平面上的协方差矩阵的元素
//     float a = cov_transformed[0][0];
//     float b = cov_transformed[0][1];
//     float c = cov_transformed[1][1];
//
//     // 计算二维协方差矩阵的行列式
//     float det = a * c - b * b;
//     float inv_det = 1.0f / det;
//
//     // 计算特征值
//     float trace = a + c;
//     float lambda_use = (trace + sqrtf(trace * trace - 4.0f * det)) / 2.0f;
//
//     // 构建新的 3x3 协方差矩阵，使用 Schur 补
//     glm::mat3 cov_new_3d(
//         c * inv_det, -b * inv_det, 0.0f,
//         -b * inv_det, a * inv_det, 0.0f,
//         0.0f, 0.0f, 1.0f / (100.0f * lambda_use));
//
//     // 将结果转换回原始坐标系
//     cov_new_3d = basis * cov_new_3d * R_transform;
//
//     // 将新的协方差矩阵的结果存储在 cov_new_small 中
//     cov_new_small[0] = cov_new_3d[0][0];
//     cov_new_small[1] = cov_new_3d[0][1];
//     cov_new_small[2] = cov_new_3d[0][2];
//     cov_new_small[3] = cov_new_3d[1][1];
//     cov_new_small[4] = cov_new_3d[1][2];
//     cov_new_small[5] = cov_new_3d[2][2];
// }

// __device__ void calculate_small(float6 cov_input, const float* viewmatrix, float3 normal, float* cov_new_small, const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy) {
//     // 如果法向量 n 的 x 和 y 分量都为 0，生成标准基向量
//     glm::vec3 v1, v2;
//
//     glm::vec3 n(normal.x, normal.y, normal.z);
//     n = glm::normalize(n);
//
// //     glm::mat3 W = glm::mat3(
// //         viewmatrix[0], viewmatrix[4], viewmatrix[8],
// //         viewmatrix[1], viewmatrix[5], viewmatrix[9],
// //         viewmatrix[2], viewmatrix[6], viewmatrix[10]);
//
//     // 声明一个 3x3 的 float 类型的协方差矩阵
//     glm::mat3 cov;
//     cov[0][0] = cov_input.x; // cov(0,0)
//     cov[0][1] = cov[1][0] = cov_input.y; // cov(0,1) and cov(1,0)
//     cov[0][2] = cov[2][0] = cov_input.w; // cov(0,2) and cov(2,0)
//     cov[1][1] = cov_input.z; // cov(1,1)
//     cov[1][2] = cov[2][1] = cov_input.u; // cov(1,2) and cov(2,1)
//     cov[2][2] = cov_input.v; // cov(2,2)
//
// //     cov = glm::transpose(W) * cov * W;
//
//     // 如果 n 的 x 和 y 分量都为 0，则沿 z 轴对齐
//     if (n.x == 0.0f && n.y == 0.0f) {
//         v1 = glm::vec3(1.0f, 0.0f, 0.0f);
//         v2 = glm::vec3(0.0f, 1.0f, 0.0f);
//         //n = glm::normalize(glm::vec3(0.0f, 0.0f, n.z));  // 保证法向量沿 z 轴方向
//     } else {
//         // 归一化法向量 n
//         // 生成与 n 正交的 v1 和 v2 向量
//         v1 = glm::normalize(glm::vec3(n.y, -n.x, 0.0f));  // 创建垂直于 n 的 v1
//         v2 = glm::normalize(glm::cross(n, v1));            // 通过叉乘计算 v2
//     }
// //     if (fabsf(glm::dot(v1, v2))>0.00001f ||fabsf(glm::dot(n, v2))>0.00001f||fabsf(glm::dot(n, v1))>0.00001f) printf("n not correct is：[%f,%f,%f]\n",n.x, n.y, n.z);
//
//     // 构建基变换矩阵
//     glm::mat3 basis(v1, v2, n);
//     glm::mat3 R_transform = glm::transpose(basis);  // R_transform 是基变换矩阵的转置
//
//     // 将输入协方差矩阵从 float* 转换为 glm::mat3
// //     glm::mat3 cov_mat(
// //         cov[0], cov[1], cov[2],
// //         cov[3], cov[4], cov[5],
// //         cov[6], cov[7], cov[8]);
//
//     // 对协方差矩阵进行基变换
//     glm::mat3 cov_transformed = R_transform * cov * basis;
//
//     // 提取二维平面上的协方差矩阵的元素
//     float a = cov_transformed[0][0];
//     float b = cov_transformed[0][1];
//     float c = cov_transformed[1][1];
//     float d = cov_transformed[0][2];
//     float e = cov_transformed[1][2];
//     float f = cov_transformed[2][2];
//
//     float div = 1/f;
//     // 构建新的 3x3 协方差矩阵，使用 Schur 补
//     glm::mat3 cov_new_3d(
//         a-d*(d*div), b-d*(e*div), 0.0f,
//         b-d*(e*div), c-e*(e*div), 0.0f,
//         0.0f, 0.0f, min(a-d*(d*div), c-e*(e*div)) / (100.0f));
//
//     // 将结果转换回原始坐标系
//     cov_new_3d = basis * cov_new_3d * R_transform;
//
//
//     float3 t = transformPoint4x3(mean, viewmatrix);
//
// 	const float limx = 1.3f * tan_fovx;
// 	const float limy = 1.3f * tan_fovy;
// 	const float txtz = t.x / t.z;
// 	const float tytz = t.y / t.z;
// 	t.x = min(limx, max(-limx, txtz)) * t.z;
// 	t.y = min(limy, max(-limy, tytz)) * t.z;
//
// 	glm::mat3 J = glm::mat3(
// 		(focal_x / t.z), 0.0, (-(focal_x * t.x) / (t.z * t.z)),
// 		0.0, (focal_y / t.z), (-(focal_y * t.y) / (t.z * t.z)),
// 		0, 0, (focal_y / t.z));
//
// 	cov_new_3d = glm::transpose(J)*cov_new_3d*J;
//     // 将新的协方差矩阵的结果存储在 cov_new_small 中
//     cov_new_small[0] = cov_new_3d[0][0];
//     cov_new_small[1] = cov_new_3d[0][1];
//     cov_new_small[2] = cov_new_3d[0][2];
//     cov_new_small[3] = cov_new_3d[1][1];
//     cov_new_small[4] = cov_new_3d[1][2];
//     cov_new_small[5] = cov_new_3d[2][2];
// }

__device__ void calculate_small(float6 cov_input, const float* viewmatrix, float3 normal, float* cov_new_small) {
    // 如果法向量 n 的 x 和 y 分量都为 0，生成标准基向量
    glm::vec3 v1, v2;

    glm::vec3 n(normal.x, normal.y, normal.z);
    n = glm::normalize(n);

//     glm::mat3 W = glm::mat3(
//         viewmatrix[0], viewmatrix[4], viewmatrix[8],
//         viewmatrix[1], viewmatrix[5], viewmatrix[9],
//         viewmatrix[2], viewmatrix[6], viewmatrix[10]);

    // 声明一个 3x3 的 float 类型的协方差矩阵
    glm::mat3 cov;
    cov[0][0] = cov_input.x; // cov(0,0)
    cov[0][1] = cov[1][0] = cov_input.y; // cov(0,1) and cov(1,0)
    cov[0][2] = cov[2][0] = cov_input.w; // cov(0,2) and cov(2,0)
    cov[1][1] = cov_input.z; // cov(1,1)
    cov[1][2] = cov[2][1] = cov_input.u; // cov(1,2) and cov(2,1)
    cov[2][2] = cov_input.v; // cov(2,2)

//     cov = glm::transpose(W) * cov * W;

    // 如果 n 的 x 和 y 分量都为 0，则沿 z 轴对齐
    if (n.x == 0.0f && n.y == 0.0f) {
        v1 = glm::vec3(1.0f, 0.0f, 0.0f);
        v2 = glm::vec3(0.0f, 1.0f, 0.0f);
        //n = glm::normalize(glm::vec3(0.0f, 0.0f, n.z));  // 保证法向量沿 z 轴方向
    } else {
        // 归一化法向量 n
        // 生成与 n 正交的 v1 和 v2 向量
        v1 = glm::normalize(glm::vec3(n.y, -n.x, 0.0f));  // 创建垂直于 n 的 v1
        v2 = glm::normalize(glm::cross(n, v1));            // 通过叉乘计算 v2
    }
//     if (fabsf(glm::dot(v1, v2))>0.00001f ||fabsf(glm::dot(n, v2))>0.00001f||fabsf(glm::dot(n, v1))>0.00001f) printf("n not correct is：[%f,%f,%f]\n",n.x, n.y, n.z);

    // 构建基变换矩阵
    glm::mat3 basis(v1, v2, n);
    glm::mat3 R_transform = glm::transpose(basis);  // R_transform 是基变换矩阵的转置

    // 将输入协方差矩阵从 float* 转换为 glm::mat3
//     glm::mat3 cov_mat(
//         cov[0], cov[1], cov[2],
//         cov[3], cov[4], cov[5],
//         cov[6], cov[7], cov[8]);

    // 对协方差矩阵进行基变换
    glm::mat3 cov_transformed = R_transform * cov * basis;

    // 提取二维平面上的协方差矩阵的元素
    float a = cov_transformed[0][0];
    float b = cov_transformed[0][1];
    float c = cov_transformed[1][1];
    float d = cov_transformed[0][2];
    float e = cov_transformed[1][2];
    float f = cov_transformed[2][2];

    float div = 1/f;

    //float lambda = 0.5f*((a-d*(d*div) + c-e*(e*div)) - sqrtf((a-d*(d*div) - c-e*(e*div))*(a-d*(d*div) + c-e*(e*div)) + 4.f*(b-d*(e*div))*(b-d*(e*div))));
    // 构建新的 3x3 协方差矩阵，使用 Schur 补
    glm::mat3 cov_new_3d(
        a-d*(d*div), b-d*(e*div), 0.0f,
        b-d*(e*div), c-e*(e*div), 0.0f,
        0.0f, 0.0f, max(a-d*(d*div), c-e*(e*div)) / (100.0f));
    //0.0f, 0.0f, lambda / (100.0f));
    // 将结果转换回原始坐标系
    cov_new_3d = basis * cov_new_3d * R_transform;

    // 将新的协方差矩阵的结果存储在 cov_new_small 中
    cov_new_small[0] = cov_new_3d[0][0];
    cov_new_small[1] = cov_new_3d[0][1];
    cov_new_small[2] = cov_new_3d[0][2];
    cov_new_small[3] = cov_new_3d[1][1];
    cov_new_small[4] = cov_new_3d[1][2];
    cov_new_small[5] = cov_new_3d[2][2];
}


// __device__ void calculate_small(float6 cov_input, const float* viewmatrix, float3 normal, float* cov_new_small) {
//     // 使用双精度类型进行中间计算
//     glm::dvec3 v1, v2;
//     glm::dvec3 n((double)normal.x, (double)normal.y, (double)normal.z);
//     n = glm::normalize(n);
//
//     glm::mat3x3 W = glm::mat3x3(
//         (double)viewmatrix[0], (double)viewmatrix[4], (double)viewmatrix[8],
//         (double)viewmatrix[1], (double)viewmatrix[5], (double)viewmatrix[9],
//         (double)viewmatrix[2], (double)viewmatrix[6], (double)viewmatrix[10]);
//
//     // 声明一个 3x3 的双精度协方差矩阵
//     glm::mat3x3 cov;
//     cov[0][0] = (double)cov_input.x; // cov(0,0)
//     cov[0][1] = cov[1][0] = (double)cov_input.y; // cov(0,1) and cov(1,0)
//     cov[0][2] = cov[2][0] = (double)cov_input.z; // cov(0,2) and cov(2,0)
//     cov[1][1] = (double)cov_input.w; // cov(1,1)
//     cov[1][2] = cov[2][1] = (double)cov_input.u; // cov(1,2) and cov(2,1)
//     cov[2][2] = (double)cov_input.v; // cov(2,2)
//
//     // 矩阵乘法，W 的转置乘 cov，再乘 W
//     cov = glm::transpose(W) * cov * W;
//
//     // 如果法向量的 x 和 y 分量为 0，设置标准基向量
//     if (n.x == 0.0 && n.y == 0.0) {
//         v1 = glm::dvec3(1.0, 0.0, 0.0);
//         v2 = glm::dvec3(0.0, 1.0, 0.0);
//     } else {
//         // 生成与 n 正交的向量 v1 和 v2
//         v1 = glm::normalize(glm::dvec3(n.y, -n.x, 0.0));
//         v2 = glm::normalize(glm::cross(n, v1));
//     }
//
//     // 基变换矩阵
//     glm::mat3x3 basis(v1, v2, n);
//     glm::mat3x3 R_transform = glm::transpose(basis);
//
//     // 变换协方差矩阵
//     glm::mat3x3 cov_transformed = R_transform * cov * basis;
//
//     // 提取二维平面上的协方差矩阵的元素
//     double a = cov_transformed[0][0];
//     double b = cov_transformed[0][1];
//     double c = cov_transformed[1][1];
//
//     // 计算二维协方差矩阵的行列式
//     double det = a * c - b * b;
//     double inv_det = 1.0 / det;
//
//     // 计算特征值
//     double trace = a + c;
//     double lambda_use = (trace + sqrt(trace * trace - 4.0 * det)) / 2.0;
//
//     // 构建新的 3x3 协方差矩阵，使用 Schur 补
//     glm::mat3x3 cov_new_3d(
//         c * inv_det, -b * inv_det, 0.0,
//         -b * inv_det, a * inv_det, 0.0,
//         0.0, 0.0, 1.0 / (1000.0 * lambda_use));
//
//     // 将结果转换回原始坐标系
//     cov_new_3d = basis * cov_new_3d * R_transform;
//
//     // 将新的协方差矩阵的结果存储在 cov_new_small 中
//     cov_new_small[0] = (float)cov_new_3d[0][0];
//     cov_new_small[1] = (float)cov_new_3d[0][1];
//     cov_new_small[2] = (float)cov_new_3d[0][2];
//     cov_new_small[3] = (float)cov_new_3d[1][1];
//     cov_new_small[4] = (float)cov_new_3d[1][2];
//     cov_new_small[5] = (float)cov_new_3d[2][2];
// }


// __device__ void calculate_small(float6 cov_input, const float* viewmatrix, float3 n, float* cov_new_small) {
//     // 使用 float 类型计算
//     glm::vec3 normal(n.x, n.y, n.z);
//     normal = glm::normalize(normal);
//
//     // 使用 float 类型的 view matrix
//     glm::mat3 W = glm::mat3(
//         viewmatrix[0], viewmatrix[4], viewmatrix[8],
//         viewmatrix[1], viewmatrix[5], viewmatrix[9],
//         viewmatrix[2], viewmatrix[6], viewmatrix[10]);
//
//     // 声明一个 3x3 的 float 类型的协方差矩阵
//     glm::mat3 cov;
//     cov[0][0] = cov_input.x; // cov(0,0)
//     cov[0][1] = cov[1][0] = cov_input.y; // cov(0,1) and cov(1,0)
//     cov[0][2] = cov[2][0] = cov_input.z; // cov(0,2) and cov(2,0)
//     cov[1][1] = cov_input.w; // cov(1,1)
//     cov[1][2] = cov[2][1] = cov_input.u; // cov(1,2) and cov(2,1)
//     cov[2][2] = cov_input.v; // cov(2,2)
//
//     cov = glm::transpose(W) * cov * W;
//
//     // 使用 float 类型的旋转矩阵
//     glm::vec3 v1, v2;
//
//     if (normal.x == 0.0f && normal.y == 0.0f) {
//         // 如果 n.x 和 n.y 为 0，设置 v1 和 v2
//         v1 = glm::vec3(1.0f, 0.0f, 0.0f);
//         v2 = glm::vec3(0.0f, 1.0f, 0.0f);
//     } else {
//         // 正交化得到 v1 和 v2
//         v1 = glm::normalize(glm::vec3(normal.y, -normal.x, 0.0f));
//         v2 = glm::normalize(glm::cross(normal, v1));
//     }
//
//     glm::mat3 R_transform(v1, v2, normal);
//     printf("R:%f,%f,%f, v10:%f, v20:%f,v30%f\n",R_transform[0][0],R_transform[0][1],R_transform[0][2],v1[0],v2[0],normal[0]);
//     glm::mat3 R_transform_T = glm::transpose(R_transform);
//
//     // 计算 cov_transformed 和 cov_inv2，使用 float 类型
//     glm::mat3 cov_transformed = R_transform_T * cov * R_transform;
//     glm::mat2 twoD_mat(cov_transformed[0][0], cov_transformed[0][1], cov_transformed[1][0], cov_transformed[1][1]);
//
//     glm::vec2 eig_val;
//     glm::mat2 eig_vec;
//
//     // 计算 2x2 矩阵的特征值
//     float trace = twoD_mat[0][0] + twoD_mat[1][1];
//     float det = twoD_mat[0][0] * twoD_mat[1][1] - twoD_mat[0][1] * twoD_mat[1][0];
//     float disc = sqrtf(trace * trace - 4.0f * det);
//
//     eig_val[0] = (trace + disc) / 2.0f;
//     eig_val[1] = (trace - disc) / 2.0f;
//
//     // 计算特征向量
//     eig_vec[0][0] = (twoD_mat[0][0] - eig_val[0]) != 0.0f ? -(twoD_mat[0][1] / (twoD_mat[0][0] - eig_val[0])) : 1.0f;
//     eig_vec[1][0] = (twoD_mat[0][0] - eig_val[0]) != 0.0f ? 1.0f : 0.0f;
//
//     eig_vec[0][1] = (twoD_mat[0][0] - eig_val[1]) != 0.0f ? -(twoD_mat[0][1] / (twoD_mat[0][0] - eig_val[1])) : 1.0f;
//     eig_vec[1][1] = (twoD_mat[0][0] - eig_val[1]) != 0.0f ? 1.0f : 0.0f;
//
//     // 归一化特征向量
//     eig_vec[1][0] = 1.0f / sqrtf(eig_vec[0][0] * eig_vec[0][0] + 1.0f);
//     eig_vec[0][0] *= eig_vec[1][0];
//     eig_vec[1][1] = 1.0f / sqrtf(eig_vec[0][1] * eig_vec[0][1] + 1.0f);
//     eig_vec[0][1] *= eig_vec[1][1];
//
//     // 将特征向量转换为 3D
//     glm::vec3 v1_2d(eig_vec[0][0], eig_vec[1][0], 0.0f);
//     glm::vec3 v2_2d(eig_vec[0][1], eig_vec[1][1], 0.0f);
//
//     // 生成 3D 特征向量矩阵
//     glm::mat3 v_3d(v1_2d, v2_2d, glm::vec3(0.0f, 0.0f, 1.0f));
//
//     // 构建特征值矩阵
//     glm::mat3 lam_mat(0.0f);
//     lam_mat[0][0] = 1.0f / eig_val[0];
//     lam_mat[1][1] = 1.0f / eig_val[1];
//     lam_mat[2][2] = 1.0f / (1000.0f * fmaxf(eig_val[0], eig_val[1]));
//
//     // 计算新的协方差矩阵 cov_new_3d
//     glm::mat3 cov_new_3d = v_3d * lam_mat * glm::transpose(v_3d);
//
//     // 将其转换回摄像机坐标系
//     cov_new_3d = R_transform * cov_new_3d * R_transform_T;
//
//     // 最终结果存储为 float32
//     cov_new_small[0] = cov_new_3d[0][0];
//     cov_new_small[1] = cov_new_3d[0][1];
//     cov_new_small[2] = cov_new_3d[0][2];
//     cov_new_small[3] = cov_new_3d[1][1];
//     cov_new_small[4] = cov_new_3d[1][2];
//     cov_new_small[5] = cov_new_3d[2][2];
// }


//high precision
// __device__ void calculate_small(float6 cov_input, const float* viewmatrix, float3 n, float* cov_new_small) {
//     // 将输入的 float 类型转换为 double 类型进行运算
//     glm::dvec3 normal((double)n.x, (double)n.y, (double)n.z);
//
//     normal = glm::normalize(normal);
//
//     glm::dmat3 W = glm::dmat3(
// 		(double)(viewmatrix[0]), (double)(viewmatrix[4]), (double)(viewmatrix[8]),
// 		(double)(viewmatrix[1]), (double)(viewmatrix[5]), (double)(viewmatrix[9]),
// 		(double)(viewmatrix[2]), (double)(viewmatrix[6]), (double)(viewmatrix[10]));
//
// //     normal[0] = normal[0]/(sqrt(normal[0]*normal[0]+normal[1]*normal[1]+normal[2]*normal[2]));
// //     normal[1] = normal[1]/(sqrt(normal[0]*normal[0]+normal[1]*normal[1]+normal[2]*normal[2]));
// //     normal[2] = normal[2]/(sqrt(normal[0]*normal[0]+normal[1]*normal[1]+normal[2]*normal[2]));
//
//     // 声明一个 3x3 的 double 类型的协方差矩阵
//     glm::dmat3 cov;
//     cov[0][0] = (double)cov_input.x; // cov(0,0)
//     cov[0][1] = cov[1][0] = (double)cov_input.y; // cov(0,1) and cov(1,0)
//     cov[0][2] = cov[2][0] = (double)cov_input.z; // cov(0,2) and cov(2,0)
//     cov[1][1] = (double)cov_input.w; // cov(1,1)
//     cov[1][2] = cov[2][1] = (double)cov_input.u; // cov(1,2) and cov(2,1)
//     cov[2][2] = (double)cov_input.v; // cov(2,2)
//
// //     printf("%f,%f,%f,%f,%f,%f\n",cov_input.x,cov_input.y,cov_input.z,cov_input.w,cov_input.u,cov_input.v);
//
//     cov=glm::transpose(W)*cov*W;
// //     cov=W*cov*glm::transpose(W);
//
//     // 使用 double 类型的旋转矩阵
//     glm::dvec3 v1, v2;
//
//     if (normal.x == 0.0 && normal.y == 0.0) {
//         // if n.x and n.y are 0, set v1 and v2
//         v1 = glm::dvec3(1.0, 0.0, 0.0);
//         v2 = glm::dvec3(0.0, 1.0, 0.0);
//     } else {
//         // 正交化得到 v1 和 v2
// //         v1 = glm::dvec3(normal.y/(sqrtf(normal.y*normal.y+normal.x*normal.x)), -normal.x/(sqrtf(normal.y*normal.y+normal.x*normal.x)), 0.0);
//         v1 = glm::normalize(glm::dvec3(normal.y, -normal.x,0.0));
//         v2 = glm::normalize(glm::cross(normal, v1));
//     }
// //     printf("\n\n%f,%f,%f,%f,%f,%f\n\n", glm::dot(v1, v2), glm::dot(normal, v2), glm::dot(v1, normal), glm::dot(v1, v1), glm::dot(v2, v2), glm::dot(normal, normal));
// //     printf("\n\n%f\n\n", v1*v2);
//
//     glm::dmat3 R_transform(v1, v2, normal);
//     glm::dmat3 R_transform_T = glm::transpose(R_transform);
//
//     // 计算 cov_transformed 和 cov_inv2，使用双精度类型
//     glm::dmat3 cov_transformed = R_transform_T * cov * R_transform;
//     glm::dmat2 twoD_mat(cov_transformed[0][0], cov_transformed[0][1], cov_transformed[1][0], cov_transformed[1][1]);
// //     glm::dmat3 cov_inv2 = glm::inverse(cov_transformed);
// //     double a = cov_transformed[0][0];  // cov_transformed(0,0)
// //     double b = cov_transformed[0][1];  // cov_transformed(0,1) == cov_transformed(1,0)
// //     double c = cov_transformed[1][1];  // cov_transformed(1,1)
// //     double d = cov_transformed[0][2];  // cov_transformed(0,2) == cov_transformed(2,0)
// //     double e = cov_transformed[1][2];  // cov_transformed(1,2) == cov_transformed(2,1)
// //     double f = cov_transformed[2][2];  // cov_transformed(2,2)
// //
// //     // Schur补公式: 计算 A_{11} - B * C^{-1} * B^T
// //     double inv_f = 1.0 / f;  // C的逆: 1/f
// //     double B_C_inv_B_T_00 = (d * d) * inv_f;
// //     double B_C_inv_B_T_01 = (d * e) * inv_f;
// //     double B_C_inv_B_T_11 = (e * e) * inv_f;
// //
// //     // A_{11} - B * C^{-1} * B^T
// //     double A11_00 = a - B_C_inv_B_T_00;
// //     double A11_01 = b - B_C_inv_B_T_01;
// //     double A11_11 = c - B_C_inv_B_T_11;
// //
// //     // 计算2x2矩阵的行列式
// //     double det_A11 = A11_00 * A11_11 - A11_01 * A11_01;
// //
// //     // 计算逆矩阵的四个元素
// //     double inv_A11_00 = A11_11 / det_A11;
// //     double inv_A11_01 = -A11_01 / det_A11;
// //     double inv_A11_11 = A11_00 / det_A11;
// //
// //     glm::dmat2 twoD_mat(inv_A11_00, inv_A11_01, inv_A11_01, inv_A11_11);
//     // 将 cov_inv2 转换为 2x2 矩阵，用于计算特征值和特征向量
// //     glm::dmat2 twoD_mat(cov_inv2[0][0], cov_inv2[0][1], cov_inv2[1][0], cov_inv2[1][1]);
//
//     glm::dvec2 eig_val;
//     glm::dmat2 eig_vec;
//
//     // 计算 2x2 矩阵的特征值
//     double trace = twoD_mat[0][0] + twoD_mat[1][1];
//     double det = twoD_mat[0][0] * twoD_mat[1][1] - twoD_mat[0][1] * twoD_mat[1][0];
//     double disc = sqrt(trace * trace - 4 * det);
//
//     eig_val[0] = (trace + disc) / 2.0;
//     eig_val[1] = (trace - disc) / 2.0;
//
//     // 计算特征向量
//     eig_vec[0][0] = (twoD_mat[0][0] - eig_val[0]) != 0 ? -(twoD_mat[0][1] / (twoD_mat[0][0] - eig_val[0])) : 1.0;
//     eig_vec[1][0] = (twoD_mat[0][0] - eig_val[0]) != 0 ? 1.0 : 0.0;
//
//     eig_vec[0][1] = (twoD_mat[0][0] - eig_val[1]) != 0 ? -(twoD_mat[0][1] / (twoD_mat[0][0] - eig_val[1])) : 1.0;
//     eig_vec[1][1] = (twoD_mat[0][0] - eig_val[1]) != 0 ? 1.0 : 0.0;
//
//     // 归一化特征向量
//     eig_vec[1][0] = 1 / sqrt(eig_vec[0][0] * eig_vec[0][0] + 1.0);
//     eig_vec[0][0] *= eig_vec[1][0];
//     eig_vec[1][1] = 1 / sqrt(eig_vec[0][1] * eig_vec[0][1] + 1.0);
//     eig_vec[0][1] *= eig_vec[1][1];
//
//     // 将特征向量转换为 3D
//     glm::dvec3 v1_2d(eig_vec[0][0], eig_vec[1][0], 0.0);
//     glm::dvec3 v2_2d(eig_vec[0][1], eig_vec[1][1], 0.0);
//
//     // 生成 3D 特征向量矩阵
//     glm::dmat3 v_3d(v1_2d, v2_2d, glm::dvec3(0.0, 0.0, 1.0));
//
//     // 构建特征值矩阵
//     glm::dmat3 lam_mat(0.0);
//     lam_mat[0][0] = 1/eig_val[0];
//     lam_mat[1][1] = 1/eig_val[1];
//     lam_mat[2][2] = 1/(1000.0 * fmax(eig_val[0], eig_val[1]));
//
//     // 计算新的协方差矩阵 cov_new_3d
//     glm::dmat3 cov_new_3d = v_3d * lam_mat * glm::transpose(v_3d);//glm::inverse(glm::transpose(v_3d) * lam_mat * v_3d);
//
//     // 将其转换回摄像机坐标系
//     cov_new_3d = R_transform * cov_new_3d * R_transform_T;
//
//     // 最终结果存储为 float32（将 double 转换为 float）
//     cov_new_small[0] = (float)cov_new_3d[0][0];
//     cov_new_small[1] = (float)cov_new_3d[0][1];
//     cov_new_small[2] = (float)cov_new_3d[0][2];
//     cov_new_small[3] = (float)cov_new_3d[1][1];
//     cov_new_small[4] = (float)cov_new_3d[1][2];
//     cov_new_small[5] = (float)cov_new_3d[2][2];
// }

// __device__  float4 generate_J(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy,const float*  viewmatrix){
//     float3 t = transformPoint4x3(mean, viewmatrix);
//
// 	const float limx = 1.3f * tan_fovx;
// 	const float limy = 1.3f * tan_fovy;
// 	const float txtz = t.x / t.z;
// 	const float tytz = t.y / t.z;
// 	t.x = min(limx, max(-limx, txtz)) * t.z;
// 	t.y = min(limy, max(-limy, tytz)) * t.z;
//
// 	glm::mat3 J = glm::mat3(
// 		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
// 		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
// 		0, 0, 0);
// 	return {J[0][0],J[0][2],J[1][1],J[1][2]};
//  }
//
//
// // //////////////////////
// // 定义一个全局变量作为 mutex
// __device__ int mutex = 0;
//
// // 函数尝试获取 mutex
// __device__ void lock(int *mutex) {
//     while (atomicCAS(mutex, 0, 1) != 0);
// }
//
// // 函数释放 mutex
// __device__ void unlock(int *mutex) {
//     atomicExch(mutex, 0);
// }
// //////////////////////

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessCUDA(int P, int D, int M,
	const float* orig_points,
	const float* normal,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* cov3D_precomp_small,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	int* radii,
	float2* points_xy_image,
	float* depths,
	float* cov3Ds,
	float* cov3D_smalls,
	float* rgb,
	float4* conic_opacity1,
	float4* conic_opacity2,
	uint4* conic_opacity3,
	float4* conic_opacity4,
	float3* conic_opacity5,
	uint4* conic_opacity6,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered,
	float3* save_normal)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;

	// Transform point by projecting
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	float3 gs_normal = {normal[3*idx],normal[3*idx+1],normal[3*idx+2]};  // is this in world coordinate or camera coordiante

    // map the normal to ray space
    //printf("%f,%f,%f\n",viewmatrix[12],viewmatrix[13],viewmatrix[14]);
	gs_normal = transformPoint4x3(gs_normal, viewmatrix);   //transform normal to camera coordinate
// 	gs_normal = transformVec4x3(gs_normal, viewmatrix);

    float sign_term = (gs_normal.z+0.00000001f)/(fabsf(gs_normal.z)+0.00000001f);

//     gs_normal = transformVec4x3(gs_normal, viewmatrix);
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	//save_normal[idx] = gs_normal;

	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters. 
// 	const float* cov3D;
    const float* cov3D;
	//float6 cov3D_inv;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 6;
	}
	else
	{
// 		cov3D_inv = computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
        computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}

    float3 cov;
    float6 view_cov3D;
	float6 cov_temp = computeCov2D_halfgs(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, view_cov3D, viewmatrix);
	cov = {cov_temp.x, cov_temp.y, cov_temp.z};

	float det = cov.x * cov.z - cov.y * cov.y; //det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;
	//float det_inv = 1.f / sqrt(det);
	float det_inv2 = 1.f / det;

	float3 conic = { cov.z * det_inv2, -cov.y * det_inv2, cov.x * det_inv2 };//conic is the inverse of the variance matrix for 2D
    //////calculate rectangle size for 2D ellipse:
    float power = logf(256.f * max(opacities[2 * idx], opacities[2 * idx+1]));//logf(2.f) * 8.0f + logf(2.f) * log2_opacity;
    int width = (int)(1.414214f * __fsqrt_rn(cov.x * power) + 1.0f);
    int height = (int)(1.414214f * __fsqrt_rn(cov.z * power) + 1.0f);

    float3 cov_small;
    const float* cov3D_small;
//     float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
//     if (cov3D_precomp_small != nullptr && cov3D_inv.x != 0&& cov3D_inv.y != 0&& cov3D_inv.z != 0 && 2*width > BLOCK_X && 2*height > BLOCK_Y)//
//     if (cov3D_precomp_small != nullptr && ((int)((point_image.x-width)/BLOCK_X)!=(int)((point_image.x)/BLOCK_X) || (int)((point_image.y-height)/BLOCK_X)!=(int)((point_image.y)/BLOCK_Y)))//2*width > BLOCK_X && 2*height > BLOCK_Y)//
	if (cov3D_precomp_small != nullptr && (2*width > BLOCK_X || 2*height > BLOCK_Y)) //cov3D_precomp_small != nullptr &&  && (fabsf(opacities[2 * idx]-opacities[2 * idx+1])>0.004f)
	{
//         calculate_small(cov3D_inv, viewmatrix, gs_normal, cov3D_smalls + idx * 6);
        calculate_small(cov_temp, viewmatrix, gs_normal, cov3D_smalls + idx * 6);
//         calculate_small(view_cov3D, viewmatrix, gs_normal, cov3D_smalls + idx * 6, p_orig, focal_x, focal_y, tan_fovx, tan_fovy);
        cov3D_small = cov3D_smalls + idx * 6;
//         cov3D_small = cov3D_precomp_small + idx * 6;
        cov_small = {cov3D_small[0],cov3D_small[1],cov3D_small[3]};//computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D_small, viewmatrix);
//         cov_small = cov;
    }
    else{
////         printf("too small!\n");
        cov_small = cov;
    }
	// Compute 2D screen-space covariance matrix
// 	float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

//     float mid_small = 0.5f * (cov_small.x + cov_small.z);
//     float det_small = cov_small.x * cov_small.z - cov_small.y * cov_small.y;;
// 	float small_lam1 = mid_small + sqrt(max(0.1f, mid_small * mid_small - det_small));
// 	float small_lam2 = mid_small - sqrt(max(0.1f, mid_small * mid_small - det_small));
	float lam1 = cov_small.x;//cov_small.x >= cov_small.z?small_lam1:small_lam2;
	float lam2 = cov_small.z;//cov_small.x < cov_small.z?small_lam1:small_lam2;

	// Invert covariance (EWA algorithm)
//     int width_small = (int)(1.414214f * __fsqrt_rn(cov_small.x * power) + 1.0f);
//     int height_small = (int)(1.414214f * __fsqrt_rn(cov_small.z * power) + 1.0f);
    int width_small = (int)(1.414214f * __fsqrt_rn(lam1 * power) + 1.0f);
    int height_small = (int)(1.414214f * __fsqrt_rn(lam2 * power) + 1.f);

    width_small = min(width_small, width);
//     printf("%f\n",(float)height_small/(float)height);
    height_small = 2*height_small>height ? min(height_small, height):max(height_small, height);//((float)height_small/(float)height) > 0.35f ?min(height_small, height):max(height_small, height); //min(height_small, height);//
//     min(height_small, height);//
    //max(height_small, height);//
    if (width <= 0 || height <= 0){
        return;
    }

    float power2 = logf(256.f * min(opacities[2 * idx], opacities[2 * idx+1]));//logf(2.f) * 8.0f + logf(2.f) * log2_opacity;
    int width_another = (int)(1.414214f * __fsqrt_rn(cov.x * power2) + 1.0f);
    int height_another = (int)(1.414214f * __fsqrt_rn(cov.z * power2) + 1.0f);

//     int width_small_another = (int)(1.414214f * __fsqrt_rn(cov_small.x * power2) + 1.0f);
//     int height_small_another = (int)(1.414214f * __fsqrt_rn(cov_small.z * power2) + 1.0f);
    int width_small_another = (int)(1.414214f * __fsqrt_rn(lam1 * power2) + 1.0f);
    int height_small_another = (int)(1.414214f * __fsqrt_rn(lam2 * power2) + 1.f);

//     printf("%f,%f,%f,%f\n",cov.x,cov.z,lam1,lam2);

    width_small_another = min(width_small_another, width_another);
    height_small_another = 2*height_small_another>height_another?min(height_small_another, height_another):max(height_small_another, height_another);//((float)height_small_another/(float)height_another) > 0.35f ?min(height_small_another, height_another):max(height_small_another, height_another); //min(height_small_another, height_another);//
//     min(height_small_another, height_another);//
//     max(height_small_another, height_another);//
    //////
	// Compute extent in screen space (by finding eigenvalues of
	// 2D covariance matrix). Use extent to compute a bounding rectangle
	// of screen-space tiles that this Gaussian overlaps with. Quit if
	// rectangle covers 0 tiles.

	float mid = 0.5f * (cov.x + cov.z);
	float lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
	//float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
	float my_radius = ceil(3.f * sqrt(lambda1));//max(lambda1, lambda2)));
// 	float my_radius = sqrtf(max(width_small, width)*max(width_small, width)+max(height_small, height)*max(height_small, height));

	float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
	uint2 rect_min, rect_max;
	uint2 rect_min_another, rect_max_another;
	float type;

	float first_divide = 1/(1.4142135f*sqrtf(max(0.00000001f,cov_temp.v - (cov_temp.w*cov_temp.w* conic.x+2*cov_temp.w*cov_temp.u*conic.y+cov_temp.u*cov_temp.u* conic.z))));
	float x_term = cov_temp.w* conic.x + cov_temp.u* conic.y;
	float y_term = cov_temp.u* conic.z + cov_temp.w* conic.y;

	gs_normal.z = 1.f/(1.4142135f*(gs_normal.z+0.000001f)); //just in case z is 0
//     if (gs_normal.z == 0.f){gs_normal.z = 1.f/(1.4142135f*0.000001f);}
//     else{
//     gs_normal.z = 1.f/(1.4142135f*gs_normal.z); //1.4142135f*
//     }
    gs_normal.x = gs_normal.x*gs_normal.z;
    gs_normal.y = gs_normal.y*gs_normal.z;
//     generate_type(cov, opacities[2 * idx], opacities[2 * idx+1], type, gs_normal, x_term, y_term);
    generate_type(cov, cov_small, opacities[2 * idx], opacities[2 * idx+1], type, gs_normal, x_term, y_term, conic);
//     generate_type(cov, opacities[2 * idx], opacities[2 * idx+1], type, gs_normal);

// 	getRect(point_image, my_radius, my_radius, rect_min, rect_max, grid);
    //float amlp = 1.f;
    getRect_another(point_image, width, height, width_small, height_small, width_another, height_another, width_small_another, height_small_another, (int)type, rect_min, rect_max, rect_min_another, rect_max_another, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	if (colors_precomp == nullptr)
	{
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	// Store some useful helper data for the next steps.
	depths[idx] = p_view.z;
	radii[idx] = my_radius;
	points_xy_image[idx] = point_image;
	// Inverse 2D covariance and opacity neatly pack into one float4
	conic_opacity1[idx] = { conic.x, conic.y, conic.z, opacities[2 * idx]};
// 	conic_opacity2[idx] = { conic.x, conic.y, conic.z, opacities[2 * idx + 1]};
// 	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
	///for new half gaussian function


    gs_normal.x = gs_normal.x*1.4142135f*first_divide;
    gs_normal.y = gs_normal.y*1.4142135f*first_divide;
	gs_normal.z = gs_normal.z*1.4142135f*first_divide;
    save_normal[idx] = gs_normal;
    conic_opacity2[idx] = {x_term*first_divide, y_term*first_divide, first_divide, opacities[2 * idx + 1]};

    uint2 rect_min2, rect_max2;
	rect_min2.x = min(rect_min.x,rect_min_another.x);
	rect_min2.y = min(rect_min.y,rect_min_another.y);
	rect_max2.x = max(rect_max.x,rect_max_another.x);
	rect_max2.y = max(rect_max.y,rect_max_another.y);
    conic_opacity3[idx] = {rect_min2.x, rect_min2.y, rect_max2.x, rect_max2.y};
    conic_opacity4[idx] = {sign_term,0.f,0.f,0.f}; //{rect_min.x, rect_min.y, rect_max.x, rect_max.y};
    conic_opacity5[idx] = {cov_temp.w,cov_temp.u,cov_temp.v};
    //conic_opacity6[idx] = {rect_min_another.x, rect_min_another.y, rect_max_another.x, rect_max_another.y};
    tiles_touched[idx] = (rect_max2.x-rect_min2.x)*(rect_max2.y-rect_min2.y);
    //float overlap= max(0, min(rect_max.y, rect_max_another.y) - max(rect_min.y, rect_min_another.y)) * max(0, min(rect_max.x, rect_max_another.x) - max(rect_min.x, rect_min_another.x));
    //tiles_touched[idx] = (rect_max.x-rect_min.x)*(rect_max.y-rect_min.y) + (rect_max_another.x-rect_min_another.x)*(rect_max_another.y-rect_min_another.y) - overlap;//(rect_max2.x-rect_min2.x)*(rect_max2.y-rect_min2.y);

    ////////////////////////only for test
//     uint2 rect_min_test, rect_max_test;
//     getRect(point_image, width, height, rect_min_test, rect_max_test, grid);
//     float3 temp_normal = transformPoint4x3({normal[3*idx],normal[3*idx+1],normal[3*idx+2]}, viewmatrix);
//     //if(rect_min_test.x!=rect_min2.x||rect_min_test.y!=rect_min2.y||rect_max_test.x!=rect_max2.x||rect_max_test.y!=rect_max2.y)
//
// //     if((int)cov.x==22 && (int)cov.y==-31 && (int)cov.z==43){
// //     if(cov.x > 10000.f){
//     if ((rect_max_test.y-rect_min_test.y) > 2*(rect_max.y-rect_min.y) && rect_min.y != 0 && rect_max.y != 67){
// //     if (height_small/height < 0.98f){
// //     if(rect_min_test.x!=rect_min2.x||rect_min_test.y!=rect_min2.y||rect_max_test.x!=rect_max2.x||rect_max_test.y!=rect_max2.y){
// //     if(width_small == 0.f || height_small == 0.f || width_small_another == 0.f || height_small_another == 0.f){
//     float4 temp_use = generate_J(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, viewmatrix);
//
// //     const float* cov3D_small;
// 	const float* cov3D_small = cov3D_precomp_small + idx * 6;
//     glm::mat3 Vrk = glm::mat3(
// 		cov3D_small[0], cov3D_small[1], cov3D_small[2],
// 		cov3D_small[1], cov3D_small[3], cov3D_small[4],
// 		cov3D_small[2], cov3D_small[4], cov3D_small[5]);
// 	glm::vec3 normal_use(temp_normal.x, temp_normal.y, temp_normal.z);
// 	//glm::vec3 result = normal_use*Vrk;
// //     printf("cov_small_3D=np.array([[%f,%f,%f],[%f,%f,%f],[%f,%f,%f])\ncov_large_3D=np.array([[%f,%f,%f],[%f,%f,%f],[%f,%f,%f]])\nview_matrix=np.array([[%f,%f,%f],[%f,%f,%f],[%f,%f,%f])\nJ=np.array([[%f,0,%f],[0,%f,%f]])\ncov_small=np.array([[%f,%f],[%f,%f]])\ncov=np.array([[%f,%f],[%f,%f]])\nn1=%f\nn2=%f\nn3=%f\nop1=%f\nop2=%f\nrec_min=[%d, %d]\nrec_max=[%d,%d]\nrec_min_new=[%d, %d]\nrec_max_new=[%d,%d]\n#type=%f\n\n",cov3D_small[0], cov3D_small[1], cov3D_small[2],cov3D_small[1], cov3D_small[3], cov3D_small[4], cov3D_small[2], cov3D_small[4], cov3D_small[5],cov3D[0], cov3D[1], cov3D[2],cov3D[1], cov3D[3], cov3D[4], cov3D[2], cov3D[4], cov3D[5],viewmatrix[0], viewmatrix[4], viewmatrix[8],viewmatrix[1], viewmatrix[5], viewmatrix[9],viewmatrix[2], viewmatrix[6], viewmatrix[10], temp_use.x, temp_use.y, temp_use.z, temp_use.w,cov_small.x , cov_small.y, cov_small.y, cov_small.z,cov.x , cov.y, cov.y, cov.z,temp_normal.x, temp_normal.y, temp_normal.z, opacities[2 * idx], opacities[2 * idx+1],rect_min_test.x,rect_min_test.y,rect_max_test.x,rect_max_test.y,rect_min2.x,rect_min2.y,rect_max2.x,rect_max2.y,type);
//     lock(&mutex);
// //     float amp = 100.f;
//     printf("cov_small=np.array([[%.11f,%.11f],[%.11f,%.11f]])\ncov=np.array([[%.11f,%.11f],[%.11f,%.11f]])\nn1=%.11f\nn2=%.11f\nn3=%.11f\nop1=%f\nop2=%f\nrec_min=[%d, %d]\nrec_max=[%d,%d]\nrec_min_new=[%d, %d]\nrec_max_new=[%d,%d]\n#type=%f\n\n",cov_small.x , cov_small.y, cov_small.y, cov_small.z,cov.x , cov.y, cov.y, cov.z,temp_normal.x, temp_normal.y, temp_normal.z, opacities[2 * idx], opacities[2 * idx+1],rect_min_test.x,rect_min_test.y,rect_max_test.x,rect_max_test.y,rect_min2.x,rect_min2.y,rect_max2.x,rect_max2.y,type);
// //     printf("cov_small_3D=np.array([[%.11f,%.11f,%.11f],[%.11f,%.11f,%.11f],[%.11f,%.11f,%.11f]])\ncov_large_3D=np.array([[%.11f,%.11f,%.11f],[%.11f,%.11f,%.11f],[%.11f,%.11f,%.11f]])\nview_matrix=np.array([[%.12f,%.12f,%.12f],[%.12f,%.12f,%.12f],[%.12f,%.12f,%.12f]])\nJ=np.array([[%f,0,%f],[0,%f,%f],[0,0,0]])\n\n",cov3D_small[0], cov3D_small[1], cov3D_small[2],cov3D_small[1], cov3D_small[3], cov3D_small[4], cov3D_small[2], cov3D_small[4], cov3D_small[5],cov3D[0], cov3D[1], cov3D[2],cov3D[1], cov3D[3], cov3D[4], cov3D[2], cov3D[4], cov3D[5],viewmatrix[0], viewmatrix[4], viewmatrix[8],viewmatrix[1], viewmatrix[5], viewmatrix[9],viewmatrix[2], viewmatrix[6], viewmatrix[10],temp_use.x, temp_use.y, temp_use.z, temp_use.w);
//     printf("grid.x = %d\ngrid.y=%d\n", grid.x, grid.y);
//     printf("new_large=np.array([[%f,%f,%f],[%f,%f,%f],[%f,%f,%f]])\n#idx=%d\nn1_ori=%.12f\nn2_ori=%.12f\nn3_ori=%.12f\n", cov_temp.x, cov_temp.y, cov_temp.w,cov_temp.y, cov_temp.z, cov_temp.u,cov_temp.w,cov_temp.u,cov_temp.v,idx, normal[3*idx],normal[3*idx+1],normal[3*idx+2]);
//     printf("J=np.array([[%f,0,%f],[0,%f,%f]])\n",temp_use.x, temp_use.y, temp_use.z, temp_use.w);
//     //printf("cov_small_3D=np.array([[%f,%f,%f],[%f,%f,%f],[%f,%f,%f]])\ncov_large_3D=np.array([[%f,%f,%f],[%f,%f,%f],[%f,%f,%f]])\nview_matrix=np.array([[%f,%f,%f],[%f,%f,%f],[%f,%f,%f]])\nJ=np.array([[%f,0,%f],[0,%f,%f]])\n\n",cov3D_small[0], cov3D_small[1], cov3D_small[2],cov3D_small[1], cov3D_small[3], cov3D_small[4], cov3D_small[2], cov3D_small[4], cov3D_small[5],cov3D[0], cov3D[1], cov3D[2],cov3D[1], cov3D[3], cov3D[4], cov3D[2], cov3D[4], cov3D[5],viewmatrix[0], viewmatrix[4], viewmatrix[8],viewmatrix[1], viewmatrix[5], viewmatrix[9],viewmatrix[2], viewmatrix[6], viewmatrix[10], temp_use.x, temp_use.y, temp_use.z, temp_use.w);
//     printf("gs_normal=[%f,%f,%f]\ncorrect_gs_normal=[%f,%f,%f]\nconicx=%f\nconicy=%f\nconicz=%f\ncon_o2x=%f\ncon_o2y=%f\ncon_o2z=%f\n",gs_normal.x,gs_normal.y,gs_normal.z,temp_normal.x*first_divide/temp_normal.z, temp_normal.y*first_divide/temp_normal.z, first_divide/temp_normal.z,conic.x, conic.y, conic.z,x_term*first_divide, y_term*first_divide, first_divide);
//     printf("width=%d\nheight=%d\nwidth_small=%d\nheight_small=%d\nwidth_another=%d\nheight_another=%d\nwidth_small_another=%d\nheight_small_another=%d\npointx=%f\npointy=%f\n\n\n",width, height, width_small, height_small,width_another, height_another, width_small_another, height_small_another, point_image.x, point_image.y);
//     unlock(&mutex);
//     }
    ///////////////////////

}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ features,
	const float4* __restrict__ conic_opacity1,
	const float4* __restrict__ conic_opacity2,
	const float4* __restrict__ conic_opacity4,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	const float* __restrict__ bg_color,
	float* __restrict__ out_color,
	const float3* __restrict__ normal)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity1[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity2[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity4[BLOCK_SIZE];
	__shared__ float3 collected_normal[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0 };

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity1[block.thread_rank()] = conic_opacity1[coll_id];
			collected_conic_opacity2[block.thread_rank()] = conic_opacity2[coll_id];
			collected_conic_opacity4[block.thread_rank()] = conic_opacity4[coll_id];
			collected_normal[block.thread_rank()] = normal[coll_id];
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			float2 xy = collected_xy[j];

			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o1 = collected_conic_opacity1[j];  //why con_o1 have three values???
			float4 con_o2 = collected_conic_opacity2[j];
			float4 con_o4 = collected_conic_opacity4[j];
			float3 norm_use = collected_normal[j];

            float power = -0.5f * (con_o1.x * d.x * d.x + con_o1.z * d.y * d.y) - con_o1.y * d.x * d.y; //
//             if (power > 0.0f || power < -4.8f)
// 				continue;
            float mask = (power <= 0.0f && power >= -4.8f) ? 1.0f : 0.0f;
            if (mask == 0.0f) continue;

			float exp_part = exp(power); //(con_o1.x * d.x * d.x + con_o1.z * d.y * d.y) - con_o1.y * d.x * d.y);
			float tanh_part1 = 1.0f+con_o4.x*erff(((norm_use.x+con_o2.x)*d.x + (norm_use.y+con_o2.y)*d.y));

//             tanh_part1 = 1.0f+erff(((norm_use.x+con_o2.x)*d.x + (norm_use.y+con_o2.y)*d.y));
//             tanh_part1 = 1.0f+erff(((norm_use.x+con_o2.x)*d.x + (norm_use.y+con_o2.y)*d.y));

			float tanh_part2 = 2.0f-tanh_part1;//(1.0f+tanh((norm_use.x*usex + norm_use.y*usey)/(-1.4142135f*norm_use.z)));

			float alpha1 = con_o1.w * exp_part * tanh_part1;
			// x,y need to be negative for another half
			float alpha2 = con_o2.w * exp_part * tanh_part2;
			//equation 9 in half gaussian paper
			float alpha = min(0.99f,0.5f*(alpha1 + alpha2)); //0.5 is for normalize each of the distribution

// 			if (alpha < 1.0f / 255.0f)
// 				continue;
            float alpha_mask = (alpha >= 1.0f / 255.0f) ? 1.0f : 0.0f;
            if (alpha_mask == 0.0f) continue;

			float test_T = T * (1 - alpha);
// 			if (test_T < 0.0001f)
// 			{
// 				done = true;
// 				continue;
// 			}
            done = (test_T < 0.0001f) ? true : done;
            if (done) continue;

			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T;

			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
	}
}

void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float2* means2D,
	const float* colors,
	const float4* conic_opacity1,
	const float4* conic_opacity2,
	const float4* conic_opacity4,
	float* final_T,
	uint32_t* n_contrib,
	const float* bg_color,
	float* out_color,
	const float3* normal)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> > (
		ranges,
		point_list,
		W, H,
		means2D,
		colors,
		conic_opacity1,
		conic_opacity2,
		conic_opacity4,
		final_T,
		n_contrib,
		bg_color,
		out_color,
		normal);
}

void FORWARD::preprocess(int P, int D, int M,
	const float* means3D,
	const float* normal,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* cov3D_precomp_small,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	int* radii,
	float2* means2D,
	float* depths,
	float* cov3Ds,
	float* cov3D_smalls,
	float* rgb,
	float4* conic_opacity1,
	float4* conic_opacity2,
	uint4* conic_opacity3,
	float4* conic_opacity4,
	float3* conic_opacity5,
	uint4* conic_opacity6,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered,
	float3* save_normal)
{
	preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		normal,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		clamped,
		cov3D_precomp,
		cov3D_precomp_small,
		colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D,
		depths,
		cov3Ds,
		cov3D_smalls,
		rgb,
		conic_opacity1,
		conic_opacity2,
		conic_opacity3,
		conic_opacity4,
		conic_opacity5,
		conic_opacity6,
		grid,
		tiles_touched,
		prefiltered,
		save_normal
		);
}

